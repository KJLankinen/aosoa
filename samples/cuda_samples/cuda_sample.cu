/*
    aosoa
    Copyright (C) 2024  Juhana Lankinen

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "aosoa/aosoa.h"
#include "aosoa/cuda_memory_operations.h"

int main(int , char **) {
    using namespace aosoa;
    hipStream_t stream = {};
    auto result = hipStreamCreate(&stream);

    CudaMemoryOperations<false> memory_ops{
        CudaAllocator{}, CudaMemcpy<false>(stream), CudaMemset<false>(stream)};

    using Soa = StructureOfArrays<256, CudaMemoryOperations<false>,
                                  Variable<float, "foo">, Variable<int, "bar">,
                                  Variable<double, "baz">>;
    Soa soa(memory_ops, 1000);

    result = hipStreamDestroy(stream);

    return 0;
}
